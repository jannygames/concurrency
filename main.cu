#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>       // For std::min
#include "hip/hip_runtime.h"
#include ""

// If you use nlohmann::json anywhere, make sure to include it properly:
// #include "json.hpp"
// using json = nlohmann::json;

using namespace std;

#define CHUNK_SIZE 10
#define THREADS_PER_BLOCK 10

/**
 * Kernel to find the maximum value in a chunk of `numbers`.
 * Each thread compares its element and does an atomicMax with the shared device max pointer.
 */
__global__ void processMaxNumberKernel(const int* numbers, int size, int* d_max)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < size)
    {
        // Use atomicMax to update the current maximum in a thread-safe manner.
        atomicMax(d_max, numbers[idx]);
    }
}

/**
 * Allocate and fill a vector of size `count` with sequential integers [0, 1, 2, ..., count-1].
 */
vector<int> AllocateSquaredNumbers(int count)
{
    vector<int> results;
    results.reserve(count);

    for (int i = 0; i < count; i++)
    {
        results.push_back(i * i);
    }

    return results;
}

int main()
{
    int count = 100;
    int maxNumber = 0;  // Host-side running maximum

    // Allocate numbers [0..99] on the host
    vector<int> numbers = AllocateSquaredNumbers(count);

    for (int i = 0; i < count; i += CHUNK_SIZE)
    {
        int currentChunkSize = min(CHUNK_SIZE, count - i);

        // Allocate device memory for the current chunk
        int* d_results;
        hipMalloc(&d_results, currentChunkSize * sizeof(int));

        // Copy chunk from host to device
        hipMemcpy(d_results, &numbers[i],
            currentChunkSize * sizeof(int),
            hipMemcpyHostToDevice);

        // Allocate device memory for the chunk maximum and initialize it with the current host max
        int* d_chunkMax;
        hipMalloc(&d_chunkMax, sizeof(int));
        hipMemcpy(d_chunkMax, &maxNumber, sizeof(int), hipMemcpyHostToDevice);

        // Number of blocks needed
        int blocks = (currentChunkSize + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

        // Launch kernel to update d_chunkMax via atomicMax
        processMaxNumberKernel << <blocks, THREADS_PER_BLOCK >> > (d_results, currentChunkSize, d_chunkMax);
        hipDeviceSynchronize();

        // Read back the chunk's max from device
        int chunkMax = 0;
        hipMemcpy(&chunkMax, d_chunkMax, sizeof(int), hipMemcpyDeviceToHost);

        // Update global max if needed
        if (chunkMax > maxNumber)
        {
            maxNumber = chunkMax;
        }

        // Clean up device memory
        hipFree(d_results);
        hipFree(d_chunkMax);
    }

    cout << "Max number: " << maxNumber << endl;

    return 0;
}